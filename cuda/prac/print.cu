
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdio>

__global__
void add(int N, float *x, float *y)
{   
    printf("threadIdx.y: %d, threadIdx.z: %d\n", threadIdx.y, threadIdx.z);
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride) { y[i] = x[i] + y[1]; }
}

int main()
{
    int N = 1 << 20;
    float *x, *y;
    
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0;
        y[i] = 2.0;
    }

    int blockSize = 256;
    int blockNum = (N + blockSize - 1) / blockSize;
    add<<<blockNum, blockSize>>>(N, x, y);

    hipDeviceSynchronize();

    float max_error = 0.0f;
    for (int i = 0; i < N; i++) {
        max_error = fmax(max_error, y[i] - 3.0f);
    }
    std::cout << "Max Error: " << max_error << std::endl;

    hipFree(x);
    hipFree(y);
}