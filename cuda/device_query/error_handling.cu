
#include <hip/hip_runtime.h>
#include <cstdio>

__global__
void saxpy(int n, float a, float *x, float *y)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) y[i] = a * y[i] + x[i];
}

int main()
{
    int n = 1 << 20;

    float *x = new float [n];
    float *y = new float [n];

    for (int i = 0; i < n; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    float *dx, *dy;
    hipMallocManaged(&dx, n * sizeof(float));
    hipMallocManaged(&dy, n * sizeof(float));

    hipMemcpy(dx, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dy, y, n * sizeof(float), hipMemcpyHostToDevice);

    saxpy<<<(n + 255) / 256, 256>>>(n, 2.0f, dx, dy);

    hipDeviceSynchronize();
    
    hipMemcpy(y, dy, n * sizeof(float), hipMemcpyDeviceToHost);

    // Get the error variable and resets it to cudaSuccess
    hipError_t errSync = hipGetLastError();

    // Asynchronous error s which occur on the device after control
    // is returned to the host, such as out-of-bounds memory accesses, 
    // require a synchronization mechanism. Any asynchronous eror is
    // returned by cudaDeviceSynchronize().
    hipError_t errAsync = hipDeviceSynchronize();
    
    if (errSync != hipSuccess) { 
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync)); 
    }
    if (errAsync != hipSuccess) { 
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync)); 
    }

    delete [] x;
    delete [] y;
    hipFree(dx);
    hipFree(dy);
}